#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2011, Alex Krizhevsky (akrizhevsky@gmail.com)
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 *
 * - Redistributions of source code must retain the above copyright notice,
 *   this list of conditions and the following disclaimer.
 * 
 * - Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 * NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <assert.h>

#include <layer_kernels.cuh>
#include "dropc/dropc_dev.hpp"
#include "dropc/dropc_bit_dev.hpp"

/*
 * E = -log(y_t)
 * probs:           (numOut, numCases)
 * labels:          (1, numCases)
 * maxProbs:        (1, numCases)
 * labelLogProbs:   (1, numCases)   (*out)
 * correctProbs:    (1, numCases)   (*out)
 * 
 * target:          (1, numCases)
 */
__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs,
                            const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];
        
        labelLogProbs[tx] = __logf(labelp);
        
        /*
         * Compute the probability of guessing the correct case if you take the most-probable label.
         * 
         * This is done like this:
         * 
         * - If the most probable label is not equal to the true label, then the probability is zero.
         * - Otherwise, the probability is 1 / (number of labels whose probability is equal to the maximum).
         * 
         * This is certainly overkill -- in practice, it's just about impossible for two labels to get assigned
         * maximum probability. But it's a safety measure to prevent over-estimating your accuracy.
         * Though it could never happen in reality. Well it could. But it wouldn't. Cool?
         */
        if (labelp != maxp) {
            correctProbs[tx] = 0;
        } else {
            int numMax = 0;
            for (int i = 0; i < numOut; i++) {
                numMax += probs[i * numCases + tx] == maxp;
            }
            correctProbs[tx] = 1.0f / float(numMax);
        }
    }
}

/*
 * E = -log(y_t)
 * y_l:     (numOut, numCases)
 * labels:  (1, numCases)
 * 
 * dE_dy_l: (numOut, numCases)
 */
template <bool add>
__global__ void kLogregCostGrad(float* y_l, float* labels, float* dE_dy_l, const int numCases,
                                 const int numOut, const float gradCoeff) {
    const int tx = blockIdx.x * LOGREG_GRAD_THREADS_X + threadIdx.x;
    const int ty = blockIdx.y * LOGREG_GRAD_THREADS_Y + threadIdx.y;
    const int tidx = ty * numCases + tx;
    
    if (ty < numOut && tx < numCases) {
        const int label = int(labels[tx]);
        float v = gradCoeff * (label == ty);
        v = __fdividef(v, y_l[tidx]);
        if (add) {
            dE_dy_l[tidx] += v;
        } else {
            dE_dy_l[tidx] = v;
        }
    }
}

/*
 * dE_dy_l: (numOut, numCases)
 * y_l:     (numOut, numCases)
 * 
 * dE_dx_l: (numOut, numCases)
 */
template <bool add>
__global__ void kSoftmaxGrad(float* dE_dy_l, float* y_l, float* dE_dx_l, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_GRAD_THREADS_X + threadIdx.x;
    const int ty = blockIdx.y * LOGREG_GRAD_THREADS_Y + threadIdx.y;
    const int tidx = ty * numCases + tx;
    
    if (ty < numOut && tx < numCases) {
        float v = 0;
        for (int j = 0; j < numOut; j++) {
            v += dE_dy_l[j * numCases + tx] * ((j == ty) - y_l[j * numCases + tx]);
        }
        v *= y_l[tidx];
        
        if (add) {
            dE_dx_l[tidx] += v;
        } else {
            dE_dx_l[tidx] = v;
        }
    }
}

/*
 * E = -log(y_t)
 * y_l:     (numOut, numCases)
 * labels:  (1, numCases)
 * 
 * dE_dx_l: (numOut, numCases)
 */
template <bool add>
__global__ void kLogregSoftmaxGrad(float* y_l, float* labels, float* dE_dx_l, const int numCases,
                                 const int numOut, const float gradCoeff) {
    const int tx = blockIdx.x * LOGREG_GRAD_THREADS_X + threadIdx.x;
    const int ty = blockIdx.y * LOGREG_GRAD_THREADS_Y + threadIdx.y;
    const int tidx = ty * numCases + tx;
    
    if (ty < numOut && tx < numCases) {
        const int label = int(labels[tx]);
        float v = gradCoeff * ((label == ty) - y_l[tidx]);
        if (add) {
            dE_dx_l[tidx] += v;
        } else {
            dE_dx_l[tidx] = v;
        }
    }
}

template <int B_X, bool add>
__global__ void kEltwiseMaxGrad(float* actGrad, float* input, float* output, float* target,
                                const int numElements) {
    for (int i = B_X * blockIdx.x + threadIdx.x; i < numElements; i += B_X * gridDim.x) {
        if (add) {
            target[i] += actGrad[i] * (output[i] == input[i]);
        } else {
            target[i] = actGrad[i] * (output[i] == input[i]);
        }
    }
}

void computeEltwiseMaxGrad(NVMatrix& actGrad, NVMatrix& input, NVMatrix& output, NVMatrix& target, bool add) {
    assert(actGrad.isContiguous());
    assert(output.isContiguous());
    assert(input.isContiguous());
    assert(actGrad.isSameDims(input));
    assert(actGrad.isSameDims(output));
    
    dim3 blocks(DIVUP(actGrad.getNumElements(), 128));
    dim3 threads(128);
    if (add) {
        assert(actGrad.isSameDims(target));
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kEltwiseMaxGrad<128), true>, hipFuncCachePreferL1);
        kEltwiseMaxGrad<128, true><<<blocks, threads>>>(actGrad.getDevData(), input.getDevData(), output.getDevData(), target.getDevData(), actGrad.getNumElements());
    } else {
        target.resize(actGrad);
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kEltwiseMaxGrad<128), false>, hipFuncCachePreferL1);
        kEltwiseMaxGrad<128, false><<<blocks, threads>>>(actGrad.getDevData(), input.getDevData(), output.getDevData(), target.getDevData(), actGrad.getNumElements());
    }
    
    cutilCheckMsg("computeEltwiseMaxGrad: Kernel execution failed");
}

/*
 * E = -log(y_t)
 * probs:           (numOut, numCases)
 * labels:          (1, numCases)
 * maxProbs:        (1, numCases)
 * labelLogProbs:   (1, numCases)   (*out)
 * correctProbs:    (1, numCases)   (*out)
 * 
 * target:          (1, numCases)
 */
void computeLogregCost(NVMatrix& labels, NVMatrix& probs, NVMatrix& labelLogProbs_out, NVMatrix& correctProbs_out) {
    int numCases = probs.getNumCols(); 
    int numOut = probs.getNumRows(); 

    assert(labels.getNumElements() == numCases);
    assert(!labels.isTrans());
    assert(!probs.isTrans());
    assert(labels.isContiguous());
    assert(probs.isContiguous());
    
    NVMatrix& maxProbs = probs.max(0);
    
    labelLogProbs_out.resize(1, numCases);
    correctProbs_out.resize(1, numCases);
    dim3 threads(LOGREG_ERR_THREADS_X, 1);
    dim3 blocks(DIVUP(numCases, LOGREG_ERR_THREADS_X), 1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kLogregCost), hipFuncCachePreferL1);
    kLogregCost<<<blocks, threads>>>(probs.getDevData(), labels.getDevData(), maxProbs.getDevData(),
                                     labelLogProbs_out.getDevData(), correctProbs_out.getDevData(),
                                     numCases, numOut);
    cutilCheckMsg("computeLogregCost: Kernel execution failed");
//    hipDeviceSynchronize();
    delete &maxProbs;
}

void computeLogregGrad(NVMatrix& labels, NVMatrix& probs, NVMatrix& target, bool add, float coeff) {
    int numCases = probs.getLeadingDim(); 
    int numOut = probs.getFollowingDim(); 
    assert(labels.getNumElements() == numCases);
    assert(probs.isContiguous());
    assert(target.isContiguous());
    assert(labels.isContiguous());
    assert(!labels.isTrans());
    assert(!probs.isTrans());
    
    dim3 threads(LOGREG_GRAD_THREADS_X, LOGREG_GRAD_THREADS_Y);
    dim3 blocks(DIVUP(numCases, LOGREG_GRAD_THREADS_X), DIVUP(numOut, LOGREG_GRAD_THREADS_Y));
    if (!add) {
        target.resize(probs);
        kLogregCostGrad<false><<<blocks, threads>>>(probs.getDevData(), labels.getDevData(), target.getDevData(),
                                                     numCases, numOut, coeff);
    } else {
        kLogregCostGrad<true><<<blocks, threads>>>(probs.getDevData(), labels.getDevData(), target.getDevData(),
                                                     numCases, numOut, coeff);
    }

    cutilCheckMsg("computeLogregGrad: Kernel execution failed");
}

void computeSoftmaxGrad(NVMatrix& acts, NVMatrix& actsGrad, NVMatrix& target, bool add) {
    int numCases = acts.getLeadingDim();
    int numOut = acts.getFollowingDim();

    assert(acts.isSameDims(actsGrad));
    assert(acts.isContiguous());
    assert(actsGrad.isContiguous());
    assert(target.isContiguous());
    assert(acts.isTrans());
    assert(actsGrad.isTrans());

    dim3 threads(LOGREG_GRAD_THREADS_X, LOGREG_GRAD_THREADS_Y);
    dim3 blocks(DIVUP(numCases, LOGREG_GRAD_THREADS_X), DIVUP(numOut, LOGREG_GRAD_THREADS_Y));
    if (!add) {
        target.resize(acts);
        kSoftmaxGrad<false><<<blocks, threads>>>(actsGrad.getDevData(), acts.getDevData(), target.getDevData(), numCases, numOut);
    } else {
        kSoftmaxGrad<true><<<blocks, threads>>>(actsGrad.getDevData(), acts.getDevData(), target.getDevData(), numCases, numOut);
    }
    cutilCheckMsg("computeSoftmaxGrad: Kernel execution failed");
}

void computeLogregSoftmaxGrad(NVMatrix& labels, NVMatrix& probs, NVMatrix& target, bool add, float coeff) {
    int numCases = probs.getLeadingDim(); 
    int numOut = probs.getFollowingDim(); 
    assert(labels.getNumElements() == numCases);
    assert(probs.isContiguous());
    assert(target.isContiguous());
    assert(labels.isContiguous());
    assert(probs.isTrans());
    
    dim3 threads(LOGREG_GRAD_THREADS_X, LOGREG_GRAD_THREADS_Y);
    dim3 blocks(DIVUP(numCases, LOGREG_GRAD_THREADS_X), DIVUP(numOut, LOGREG_GRAD_THREADS_Y));
    if (!add) {
        target.resize(probs);
        kLogregSoftmaxGrad<false><<<blocks, threads>>>(probs.getDevData(), labels.getDevData(), target.getDevData(),
                                                     numCases, numOut, coeff);
    } else {
        kLogregSoftmaxGrad<true><<<blocks, threads>>>(probs.getDevData(), labels.getDevData(), target.getDevData(),
                                                     numCases, numOut, coeff);
    }

    cutilCheckMsg("computeLogregSoftmaxGrad: Kernel execution failed");
}

//-------------------------------------------------------
//   functions related with dropc
//-------------------------------------------------------
void computeFCDropC_fprop( 
      NVMatrix&  x,         ///<[in]  input matrix x, col major, numData x inDim
      NVMatrix&  w,         ///<[in]  weight matrix w, col major, inDim x outDim
      NVMatrix&  b,         ///<[in]  bias matrix, row major, 1 x outDim
      NVMatrix& mw,         ///<[in]  maskWeights, col major, inDim x (outDimxnumData)
      NVMatrix& mb,         ///<[in]  maskBiases, col major, dataDim x outDim          
      NVMatrix& y           ///<[in,out] target matrix y, col major, dataDim x outDim
      ){
   // pre-condition check
   assert( x.isTrans() );
   int numData = x.getNumRows();
   int inDim = x.getNumCols();

   assert( w.isTrans() );
   assert( w.getNumRows() == inDim );
   int outDim = w.getNumCols();

   assert( !b.isTrans() );
   assert( b.getNumRows() == 1 && b.getNumCols() == outDim );

   assert( mw.isTrans() );
   assert( mw.getNumRows() == inDim && mw.getNumCols() == (outDim*numData) );

   assert( mb.isTrans() );
   assert( mb.getNumRows() == numData&& mb.getNumCols() == outDim );

   assert( y.isTrans() );
   assert( y.getNumRows() == numData && y.getNumCols() == outDim );

   // call dev function
   computeFCDropC_fprop_d(
         x.getDevData(), w.getDevData(), b.getDevData(), // input matrix
         //m, n, d, // dims
         outDim, inDim, numData,
         mw.getDevData(), mb.getDevData(),  // masks
         y.getDevData()        // output
         );

}

void computeFCDropC_bpropActs(
      NVMatrix& v,         ///<[in]  bprop act from previous layer, col major,numData x outDim
      NVMatrix& w,         ///<[in]  weight matrix w, col major, inDim x outDim
      float scale_g,       ///<[in]  input gradient scale
      NVMatrix& mw,        ///<[in]  maskWeights, col major, inDim x (outDimxnumData)
      NVMatrix& da,        ///<[in,out] d-active, col major, numData x inDim              
      float scale_da       ///<[in]  da scale
      ){
   // pre-condition check
   assert( v.isTrans() );
   int numData = v.getNumRows();
   int outDim = v.getNumCols();
   
   assert( w.isTrans() );
   int inDim = w.getNumRows();
   assert( w.getNumCols() == outDim );

   assert( mw.isTrans() );
   assert( mw.getNumRows() == inDim && mw.getNumCols() == (outDim*numData) );

   assert( da.isTrans() );
   assert( da.getNumRows() == numData && da.getNumCols() == inDim );

   // call dev function
   computeFCDropC_bpropActs_d(
         v.getDevData(), w.getDevData(),
         //m, n, d,
         outDim, inDim, numData,
         scale_g,
         mw.getDevData(),
         da.getDevData(),
         scale_da 
         );
}

void computeFCDropC_bpropWeights(
      NVMatrix& a,            ///<[in] prev activation matrix, col major, numData x inDim
      NVMatrix& v,            ///<[in] gradient matrix, col major, numData x outDim
      float scale_g,          ///<[in] inc scale
      NVMatrix& mw,           ///<[in] maskWeights, col major, inDim x (outDimxnumData)
      NVMatrix& dw,           ///<[in,out] w gradient, col major, inDim x outDim
      float scale_dw          ///<[in] gradient scale
      ){
   // pre-condition check
   assert( a.isTrans() );
   int numData = a.getNumRows();
   int inDim = a.getNumCols();

   assert( v.isTrans() );
   assert( v.getNumRows() == numData );
   int outDim = v.getNumCols();

   assert( mw.isTrans() );
   assert( mw.getNumRows() == inDim && mw.getNumCols() == (outDim*numData) );

   assert( dw.isTrans() );
   assert( dw.getNumRows() == inDim && dw.getNumCols() == outDim );

   // call dev function
   computeFCDropC_bpropWeights_d(
         a.getDevData(), v.getDevData(),
         //m, n, d,
         outDim, inDim, numData,
         scale_g,
         mw.getDevData(),
         dw.getDevData(), scale_dw
         );

}

void computeFCDropC_bit_fprop( 
      NVMatrix&  x,         ///<[in]  input matrix x, col major, numData x inDim
      NVMatrix&  w,         ///<[in]  weight matrix w, col major, inDim x outDim
      NVMatrix&  b,         ///<[in]  bias matrix, row major, 1 x outDim
      const MaskWeights& mw,  ///<[in]  maskWeights object
      NVMatrix& mb,         ///<[in]  maskBiases, col major, dataDim x outDim          
      NVMatrix& y           ///<[in,out] target matrix y, col major, dataDim x outDim
      ){
   // pre-condition check
   assert( x.isTrans() );
   int numData = x.getNumRows();
   int inDim = x.getNumCols();

   assert( w.isTrans() );
   assert( w.getNumRows() == inDim );
   int outDim = w.getNumCols();

   assert( !b.isTrans() );
   assert( b.getNumRows() == 1 && b.getNumCols() == outDim );

   assert( mb.isTrans() );
   assert( mb.getNumRows() == numData&& mb.getNumCols() == outDim );

   assert( y.isTrans() );
   assert( y.getNumRows() == numData && y.getNumCols() == outDim );

   // call dev function
   computeFCDropC_bit_fprop_d(
         x.getDevData(), w.getDevData(), b.getDevData(), // input matrix
         //m, n, d, // dims
         outDim, inDim, numData,
         mw, //mask w
         mb.getDevData(),  // mask b
         y.getDevData()        // output
         );

}

void computeFCDropC_bit_bpropActs(
      NVMatrix& v,         ///<[in]  bprop act from previous layer, col major,numData x outDim
      NVMatrix& w,         ///<[in]  weight matrix w, col major, inDim x outDim
      float scale_g,       ///<[in]  input gradient scale
      const MaskWeights& mw,  ///<[in]  maskWeights object
      NVMatrix& da,        ///<[in,out] d-active, col major, numData x inDim              
      float scale_da       ///<[in]  da scale
      ){
   // pre-condition check
   assert( v.isTrans() );
   int numData = v.getNumRows();
   int outDim = v.getNumCols();
   
   assert( w.isTrans() );
   int inDim = w.getNumRows();
   assert( w.getNumCols() == outDim );

   assert( da.isTrans() );
   assert( da.getNumRows() == numData && da.getNumCols() == inDim );

   // call dev function
   computeFCDropC_bit_bpropActs_d(
         v.getDevData(), w.getDevData(),
         //m, n, d,
         outDim, inDim, numData,
         scale_g,
         mw, 
         da.getDevData(),
         scale_da 
         );
}

void computeFCDropC_bit_bpropWeights(
      NVMatrix& a,            ///<[in] prev activation matrix, col major, numData x inDim
      NVMatrix& v,            ///<[in] gradient matrix, col major, numData x outDim
      float scale_g,          ///<[in] inc scale
      const MaskWeights& mw,  ///<[in]  maskWeights object
      NVMatrix& dw,           ///<[in,out] w gradient, col major, inDim x outDim
      float scale_dw          ///<[in] gradient scale
      ){
   // pre-condition check
   assert( a.isTrans() );
   int numData = a.getNumRows();
   int inDim = a.getNumCols();

   assert( v.isTrans() );
   assert( v.getNumRows() == numData );
   int outDim = v.getNumCols();

   assert( dw.isTrans() );
   assert( dw.getNumRows() == inDim && dw.getNumCols() == outDim );

   // call dev function
   computeFCDropC_bit_bpropWeights_d(
         a.getDevData(), v.getDevData(),
         //m, n, d,
         outDim, inDim, numData,
         scale_g,
         mw,
         dw.getDevData(), scale_dw
         );

}

void computeFCDropC_bit_inference(
        NVMatrix& mu,       ///<[in]  mean matrix, col major, dataDim x outDim
        NVMatrix& var,      ///<[in]  var matrix,  col major, dataDim x outDim
        int numSamples,     ///<[in]  number of samples for mc sampling
        NVMatrix& y         ///<[in,out] target matrix y, col major, dataDim x outDim
        ){
    int numData = mu.getNumRows();
    int outDim = mu.getNumCols();
    size_t num_elements = numData * outDim;
    assert( mu.isTrans() );

    assert( var.getNumRows() == numData );
    assert( var.getNumCols() == outDim );
    assert( var.isTrans() );

    assert( y.getNumRows() == numData );
    assert( y.getNumCols() == outDim );
    assert( y.isTrans() );

    // call dev funtion
    computeFCDropC_bit_inference_d( mu.getDevData(), 
        var.getDevData(), num_elements,
        numSamples, y.getDevData());
}
